#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdlib.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <ios>
#include <iostream>
#include <random>


const std::string kFilename = "out.xyz";
const int kBlockDim = 1024;
const int kNumParticles = 10;
__device__ const float kDeltaT = 0.1;
const float kGravitationalConstant = 6.67430e-11;
const int kNumTimesteps = 100000;
const int kNumTimestepsSnapshot = 100;


struct Vec3 {
    float x;
    float y;
    float z;

    float operator[](int idx) const {
        if (idx == 0) {
            return x;
        }
        if (idx == 1) {
            return y;
        }
        return z;
    }

    __device__ Vec3 operator+(const Vec3& rhs) const {
        return {x + rhs.x, y + rhs.y, z + rhs.z};
    }

    __device__ Vec3 operator-(const Vec3& rhs) const {
        return {x - rhs.x, y - rhs.y, z - rhs.z};
    }

    __device__ Vec3 operator*(const Vec3& rhs) const {
        return {x * rhs.x, y * rhs.y, z * rhs.z};
    }

    __device__ Vec3 operator*(const float& rhs) const {
        return {x * rhs, y * rhs, z * rhs};
    }

    __device__ Vec3 operator/(const Vec3& rhs) const {
        return {x / rhs.x, y / rhs.y, z / rhs.z};
    }

    __device__ Vec3 operator/(const float& rhs) const {
        return {x / rhs, y / rhs, z / rhs};
    }

    __device__ bool operator==(const Vec3& rhs) const {
        return x == rhs.x && y == rhs.y && z == rhs.z;
    }
};


struct Particle {
    Vec3 pos;
    Vec3 vel;
    Vec3 acc;
    float mass;
    float radius;
    float potential_energy = 0;
    float kinetic_energy = 0;
};


__global__ void UpdatePosition(Particle* particles) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= kNumParticles) {
        return;
    }
    particles[index].pos = particles[index].pos + particles[index].vel * kDeltaT;
}


__global__ void UpdateVelocityHalf(Particle* particles) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= kNumParticles) {
        return;
    }
    particles[index].vel = particles[index].vel + particles[index].acc * kDeltaT / 2;
}


__device__ float GetNorm(const Vec3& vec) {
    return std::sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}


__device__ Vec3 GetUnitVector(const Vec3& vec) {
    return vec / GetNorm(vec);
}


__device__ Vec3 GravitationalForce(const Particle& first, const Particle& second) {
    Vec3 direction_unit_vector = GetUnitVector(second.pos - first.pos);
    float force = kGravitationalConstant * first.mass * second.mass / ((GetNorm(second.pos - first.pos) + 0.1) * (GetNorm(second.pos - first.pos) + 0.1));
    return direction_unit_vector * force;
}


__global__ void DeriveAcc(Particle* particles) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= kNumParticles) {
        return;
    }
    particles[index].acc = {0, 0, 0};
    for (int i = 0; i < kNumParticles; i++) {
        if (i == index) {
            continue;
        }
        particles[index].acc = particles[index].acc + GravitationalForce(particles[index], particles[i]) / particles[index].mass;
    }
}


__device__ float GetPotential(const Particle& first, const Particle& second) {
    return kGravitationalConstant * first.mass * second.mass / (GetNorm(second.pos - first.pos) + 0.1);
}


__global__ void CalculatePotential(Particle* particles) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= kNumParticles) {
        return;
    }
    particles[index].potential_energy = 0;
    for (int i = 0; i < kNumParticles; i++) {
        if (i == index) {
            continue;
        }
        particles[index].potential_energy = particles[index].potential_energy + GetPotential(particles[index], particles[i]);
    }
}


__global__ void CalculateKinetic(Particle* particles) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= kNumParticles) {
        return;
    }
    particles[index].kinetic_energy = 1. / 2 * particles[index].mass * GetNorm(particles[index].vel) * GetNorm(particles[index].vel);
}


int main() {
    {
        std::ofstream outfile;
        outfile.open(kFilename);
        outfile << kNumParticles << " " << kNumTimesteps << " " << kNumTimestepsSnapshot << "\n";
        outfile.close();
    }
    std::ofstream outfile(kFilename, std::ios_base::app);

    Particle* particles = (Particle*)malloc(sizeof(Particle) * kNumParticles);
    Particle* d_particles;

    std::default_random_engine gen;
    std::uniform_real_distribution<float> distribution_x(-100, 100);
    std::uniform_real_distribution<float> distribution_y(-100, 100);
    std::uniform_real_distribution<float> distribution_z(-2, 2); // top
    std::uniform_real_distribution<float> distribution_mass(1e9, 1e9);
    std::uniform_real_distribution<float> distributaion_vel_x(-0, 0);
    std::uniform_real_distribution<float> distributaion_vel_y(-0, 0);

    for (int i = 0; i < kNumParticles; i++) {
        Vec3 rand_pos = {distribution_x(gen), distribution_y(gen), distribution_z(gen)};
        particles[i] = {rand_pos, {distributaion_vel_x(gen), distributaion_vel_y(gen), 0}, {0, 0, 0}, distribution_mass(gen), 10};
    }

    hipMalloc((void**)&d_particles, sizeof(Particle) * kNumParticles);
    hipMemcpy(d_particles, particles, sizeof(Particle) * kNumParticles, hipMemcpyHostToDevice);

    for (int timestep = 0; timestep < kNumTimesteps; timestep++) {
        UpdateVelocityHalf<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
        hipDeviceSynchronize();
        UpdatePosition<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
        hipDeviceSynchronize();
        DeriveAcc<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
        hipDeviceSynchronize();
        UpdateVelocityHalf<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
        hipDeviceSynchronize();

        if (timestep % kNumTimestepsSnapshot == kNumTimestepsSnapshot-1) {
            CalculatePotential<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
            hipDeviceSynchronize();
            CalculateKinetic<<<(kNumParticles/kBlockDim)+1, kBlockDim>>>(d_particles);
            hipDeviceSynchronize();
            hipMemcpy(particles, d_particles, sizeof(Particle) * kNumParticles, hipMemcpyDeviceToHost);
            float potential_energy = 0;
            float kinetic_energy = 0;
            for (int i = 0; i < kNumParticles; i++) {
                potential_energy += particles[i].potential_energy / 2;
                kinetic_energy += particles[i].kinetic_energy;
            }

            std::cout << timestep << " " << potential_energy << " " << kinetic_energy << " "<< potential_energy + kinetic_energy << std::endl;
            /*for (int i = 0; i < kNumParticles; i++) {
                outfile << particles[i].pos.x << " " << particles[i].pos.y << " " << particles[i].pos.z << "\n";
            }*/
            outfile << "\n";
        }
    }
}
