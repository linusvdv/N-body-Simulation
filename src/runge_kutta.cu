#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <iostream>

#define float double

const std::string kFilename = "RKout.xyz";
const std::string kPotentialFilename = "RKout.energy";
const std::string kMomentumFilename = "RKout.momentum";
const int kBlockDim = 16;
const int kNumParticles = 3;
__device__ const float kGravitationalConstant = 6.67430e-11;
const int kNumTimestepsSnapshot = 100;
__device__ const float kDDistance = 0;
constexpr int kGridDim = ((kNumParticles-1)/kBlockDim)+1;
__device__ const double kDeltaT = 1; // s
const int kNumTimesteps = 31556950;


struct Vec3 {
    float x = 0;
    float y = 0;
    float z = 0;

    float operator[](int idx) const {
        if (idx == 0) {
            return x;
        }
        if (idx == 1) {
            return y;
        }
        return z;
    }

    __host__ __device__ Vec3 operator+(const Vec3& rhs) const {
        return {x + rhs.x, y + rhs.y, z + rhs.z};
    }

    __host__ __device__ Vec3 operator-(const Vec3& rhs) const {
        return {x - rhs.x, y - rhs.y, z - rhs.z};
    }

    __host__ __device__ Vec3 operator*(const Vec3& rhs) const {
        return {x * rhs.x, y * rhs.y, z * rhs.z};
    }

    __host__ __device__ Vec3 operator*(const float& rhs) const {
        return {x * rhs, y * rhs, z * rhs};
    }

    __host__ __device__ Vec3 operator/(const Vec3& rhs) const {
        return {x / rhs.x, y / rhs.y, z / rhs.z};
    }

    __host__ __device__ Vec3 operator/(const float& rhs) const {
        return {x / rhs, y / rhs, z / rhs};
    }

    __host__ __device__ bool operator==(const Vec3& rhs) const {
        return x == rhs.x && y == rhs.y && z == rhs.z;
    }
};


struct State {
    Vec3 pos;
    Vec3 vel;

    __host__ __device__ State operator+(const State& rhs) const {
        return {pos + rhs.pos, vel + rhs.vel};
    }
    __host__ __device__ State operator*(const float& rhs) const {
        return {pos * rhs, vel * rhs};
    }
};


struct Particle {
    State state;

    float mass;
    float radius;
    float potential_energy = 0;
    float kinetic_energy = 0;

    State ks[4 + 1] = {};  // runge kutta 4 plus 0th index
};


__host__ __device__ float GetChangedNorm(const Vec3& vec) {
    return std::sqrt((vec.x*vec.x) + (vec.y*vec.y) + (vec.z*vec.z) + (kDDistance*kDDistance));
}


__host__ __device__ float GetNorm(const Vec3& vec) {
    return std::sqrt((vec.x*vec.x) + (vec.y*vec.y) + (vec.z*vec.z));
}


__device__ Vec3 GetUnitVector(const Vec3& vec) {
    return vec / GetNorm(vec);
}


__device__ Vec3 GravitationalAcceleration(const State& first, const State& second, const float& other_mass) {
    float norm = GetChangedNorm(second.pos - first.pos);
    Vec3 force = (second.pos - first.pos) * kGravitationalConstant / (norm * norm) * other_mass / norm;
    return force;
}


__device__ float GetPotential(const Particle& first, const Particle& second) {
    return -kGravitationalConstant * first.mass / GetChangedNorm(second.state.pos - first.state.pos) * second.mass;
}


__global__ void UpdatePositionVelocity(Particle* particles) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= kNumParticles) {
        return;
    }
    particles[index].state = particles[index].state + (particles[index].ks[1]
                                                     + particles[index].ks[2] * 2
                                                     + particles[index].ks[3] * 2
                                                     + particles[index].ks[4]) * (kDeltaT / 6);
}


template<int k>
__global__ void DeriveKS(Particle* particles) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= kNumParticles) {
        return;
    }
    float devising_factor[] = {1, 2, 2, 1};
    Vec3 acc = {0, 0, 0};
    for (int i = 0; i < kNumParticles; i++) {
        if (i == index) {
            continue;
        }
        acc = acc + GravitationalAcceleration((particles[index].state + particles[index].ks[k-1] * (kDeltaT / devising_factor[k-1])),
                                              (particles[i].state + particles[i].ks[k-1] * (kDeltaT / devising_factor[k-1])),
                                              particles[i].mass);
    }
    particles[index].ks[k] = {particles[index].state.vel, acc};
}


__global__ void CalculatePotential(Particle* particles) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= kNumParticles) {
        return;
    }
    particles[index].potential_energy = 0;
    for (int i = 0; i < kNumParticles; i++) {
        if (i == index) {
            continue;
        }
        particles[index].potential_energy = particles[index].potential_energy + GetPotential(particles[index], particles[i]);
    }
}


__global__ void CalculateKinetic(Particle* particles) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= kNumParticles) {
        return;
    }
    particles[index].kinetic_energy = 1. / 2 * particles[index].mass * GetNorm(particles[index].state.vel) * GetNorm(particles[index].state.vel);
}


int main() {
    {
        std::ofstream outfile;
        outfile.open(kFilename);
        outfile << kNumParticles << " " << kNumTimesteps << " " << kNumTimestepsSnapshot << "\n";
        outfile.close();
        std::ofstream potential_file;
        std::ofstream momentum_file;
        potential_file.open(kPotentialFilename);
        momentum_file.open(kMomentumFilename);
        potential_file.close();
        momentum_file.close();
    }
    std::ofstream outfile(kFilename, std::ios_base::app);
    std::ofstream potential_file(kPotentialFilename, std::ios_base::app);
    std::ofstream momentum_file(kMomentumFilename, std::ios_base::app);

    Particle* particles = (Particle*)malloc(sizeof(Particle) * kNumParticles);
    Particle* d_particles;

    particles[0] = {{{0.000000e+00, 0.000000e+00, 0.000000e+00}, {0.000000e+00, 0.000000e+00, 0.000000e+00}}, 1.9885e+30};
    particles[1] = {{{-2.649903e+10, 1.446973e+11, -6.111494e+05}, {-2.979426e+04, -5.469295e+03, 1.817837e-01}}, 5.9722e+24};
    particles[2] = {{{-2.679064e+10, 1.444223e+11, 3.566005e+07}, {-2.915073e+04, -6.200279e+03, -1.132468e+01}}, 7.342e+22};

    hipMalloc((void**)&d_particles, sizeof(Particle) * kNumParticles);
    hipMemcpy(d_particles, particles, sizeof(Particle) * kNumParticles, hipMemcpyHostToDevice);

    // get start value
    float start_potential_energy = 0;
    float start_kinetic_energy = 0;
    {
        CalculatePotential<<<kGridDim, kBlockDim>>>(d_particles);
        CalculateKinetic<<<kGridDim, kBlockDim>>>(d_particles);
        hipMemcpy(particles, d_particles, sizeof(Particle) * kNumParticles, hipMemcpyDeviceToHost);
        for (int i = 0; i < kNumParticles; i++) {
            start_potential_energy += particles[i].potential_energy / 2;
            start_kinetic_energy += particles[i].kinetic_energy;
        }
    }

    for (int timestep = 0; timestep < kNumTimesteps; timestep++) {
        DeriveKS<1><<<kGridDim, kBlockDim>>>(d_particles);
        DeriveKS<2><<<kGridDim, kBlockDim>>>(d_particles);
        DeriveKS<3><<<kGridDim, kBlockDim>>>(d_particles);
        DeriveKS<4><<<kGridDim, kBlockDim>>>(d_particles);
        UpdatePositionVelocity<<<kGridDim, kBlockDim>>>(d_particles);

        if (timestep % kNumTimestepsSnapshot == kNumTimestepsSnapshot-1) {
            CalculatePotential<<<kGridDim, kBlockDim>>>(d_particles);
            CalculateKinetic<<<kGridDim, kBlockDim>>>(d_particles);
            hipMemcpy(particles, d_particles, sizeof(Particle) * kNumParticles, hipMemcpyDeviceToHost);
            float potential_energy = 0;
            float kinetic_energy = 0;
            Vec3 total_momentum = {0, 0, 0};
            for (int i = 0; i < kNumParticles; i++) {
                potential_energy += particles[i].potential_energy / 2;
                kinetic_energy += particles[i].kinetic_energy;
                total_momentum = total_momentum + particles[i].state.vel * particles[i].mass;
            }

            std::cout << timestep << std::endl;
            potential_file << timestep * kDeltaT << " " << potential_energy-start_potential_energy << " " << kinetic_energy-start_kinetic_energy << " "<< potential_energy-start_potential_energy + kinetic_energy-start_kinetic_energy << std::endl;
            momentum_file << timestep * kDeltaT << " " << total_momentum.x << " " << total_momentum.y << " " << total_momentum.z << " " << GetNorm(total_momentum) << std::endl;
            for (int i = 0; i < kNumParticles; i++) {
                outfile << particles[i].state.pos.x << " " << particles[i].state.pos.y << " " << particles[i].state.pos.z << "\n";
            }
            outfile << "\n";
        }
    }
}
