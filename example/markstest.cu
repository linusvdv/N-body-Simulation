#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

//copied from the internet, if x, y arrays outputs a*x + y, where a constant int

__global__
void saxpy(int n, int a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n){
    for(int j = 0; j<a; j++)y[i] = y[i] + x[i];
  }
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float)); //allocate
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); //put it in device memory
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on N elements; 
  saxpy<<<(N)/128, 128>>>(N, N, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]- ((1<<20)+2)));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
