#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

int const n = 512;
__device__ int sz = n;
#define IDX(i, j, N) ((i) * (N) + (j))

__global__ void matrix_multi(int *a, int *b, int *c){

    int row = blockIdx.x;
    int col = threadIdx.x;

    int val = 0;
    for(int i = 0; i<sz; i++){
        val += a[IDX(row, i, sz)] * b[IDX(i, col, sz)];
    }
    c[IDX(row, col, sz)] = val;
}

int main(){
    int a[n*n], b[n*n], c[n*n];
    int *d_a, *d_b, *d_c;
    int size = n * n * sizeof(int);
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    for(int i = 0; i<n; i++){
        for(int j = 0; j<n; j++){
            a[IDX(i, j, n)] = 1;
            b[IDX(i, j, n)] = 1;
            c[IDX(i, j, n)] = 3;
        }
    }
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    matrix_multi<<<n,n>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    int res = 0;
    for(int i = 0; i < n; i++){
        for(int j = 0; j<n; j++)res += c[IDX(i, j, n)];
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("%d\n", res);
    return 0;
}