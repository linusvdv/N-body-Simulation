#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void CudaHello(){
    printf("Hello World from GPU!\n");
}


int main() {
    CudaHello<<<2,2>>>(); 
    hipDeviceSynchronize();
    return 0;
}
