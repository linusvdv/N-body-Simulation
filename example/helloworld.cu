#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<2,2>>>(); 
    hipDeviceSynchronize();
    return 0;
}
