#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>


__global__ void Square(const int *cuda_input, int *cuda_output, int num_elements) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int sum = 0;
    for (int i = 0; i < num_elements; i++) {
        sum += (cuda_input[i] - cuda_input[index]) * (cuda_input[i] - cuda_input[index]);
    }
    cuda_output[index] = sum;
}


int main() {
    const int num_elements = 128*1024;
    int* cuda_input = (int *)malloc(sizeof(int)*num_elements);
    int* cuda_output = (int *)malloc(sizeof(int)*num_elements);
    int* d_cuda_input;
    int* d_cuda_output;
    srand(0);
    std::cout << "CPU: ";
    for (int i = 0; i < num_elements; i++) {
        cuda_input[i] = rand()%100;
//        std::cout << cuda_input[i] << " ";
    }
    std::cout << "\n";

    hipMalloc((void **)&d_cuda_input, sizeof(int)*num_elements);
    hipMalloc((void **)&d_cuda_output, sizeof(int)*num_elements);

    hipMemcpy(d_cuda_input, cuda_input, sizeof(int)*num_elements, hipMemcpyHostToDevice);

    Square<<<num_elements/1024, 1024>>>(d_cuda_input, d_cuda_output, num_elements);
    hipDeviceSynchronize();

    hipMemcpy(cuda_output, d_cuda_output, sizeof(int)*num_elements, hipMemcpyDeviceToHost);

    std::cout << "CPU: ";
    for (int i = 0; i < num_elements; i++) {
//        std::cout << cuda_output[i] << " ";
    }
    std::cout << "\n";

    hipFree(d_cuda_input);
    hipFree(d_cuda_output);
}
